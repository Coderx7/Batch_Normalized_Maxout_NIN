#include "hip/hip_runtime.h"
 // @file pooling_gpu.cu
// @brief Pooling block implementation (GPU)
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "maxout.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

/* ---------------------------------------------------------------- */
/*                                              pooling_max_forward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
maxout_kernel
(T* pooled,
 const T* data,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int numunit,
 const int numpiece)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    int thx = pooledIndex % (pooledWidth*pooledHeight);  // which element in pooled    
    int ut  = (pooledIndex / (pooledWidth*pooledHeight)) % numunit; //which unit
    int ntr = pooledIndex / (pooledWidth*pooledHeight*numunit); // which trial 
    
    T bestValue = data[thx + pooledWidth*pooledHeight*(ut + ntr*numunit*numpiece)];  // GET vlaue in data

        //for (int k = 0; k < numpiece ; ++k) {     
        //   bestValue = max(bestValue, data[thx + pooledWidth*pooledHeight*(ut*numpiece+k)]) ;
        // }
        for (int k = 0; k < numpiece ; ++k) {     
           bestValue = max(bestValue, data[thx + pooledWidth*pooledHeight*(ut + k*numunit + ntr*numunit*numpiece)]) ;
        }

    pooled[pooledIndex] = bestValue ;
    
  }
}

template<> vl::Error
vl::impl::maxout_forward<vl::GPU, float>(float* pooled,
                                              float const* data,
                                              size_t height, size_t width, size_t depth,
                                              size_t numunit, size_t numpiece)
{
  int pooledWidth = width;
  int pooledHeight = height;
  int pooledVolume = pooledWidth * pooledHeight * depth / numpiece ;
  
  maxout_kernel<float>
  <<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (pooled, data,
   pooledHeight, pooledWidth, pooledVolume,
   numunit, numpiece);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}


/* ---------------------------------------------------------------- */
/*                                             pooling_max_backward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
maxout_backward_kernel
(T* derData,
 const T* data,
 const T* derPooled,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int numunit,
 const int numpiece)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {

    int thx = pooledIndex % (pooledWidth*pooledHeight);  // which element in pooled
    int ut  = (pooledIndex / (pooledWidth*pooledHeight)) % numunit; //which unit
    int ntr = pooledIndex / (pooledWidth*pooledHeight*numunit); // which trial   

    //T bestValue = data[thx + pooledWidth*pooledHeight*(ut*numpiece)];  // GET vlaue in data

	T bestValue = data[thx + pooledWidth*pooledHeight*(ut +  ntr*numunit*numpiece)];  // GET vlaue in data

    int bestindex = 0;    
        for (int k = 0; k < numpiece ; ++k) {
			//T value = data[thx + pooledWidth*pooledHeight*(ut*numpiece+k)];
			T value = data[thx + pooledWidth*pooledHeight*(ut + k*numunit +  ntr*numunit*numpiece)];
              
               if (value > bestValue) {
					bestValue = value ;
					bestindex = k;    
				}
        }


    /*
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requrires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    //int dain = thx + pooledWidth*pooledHeight*(ut*numpiece+bestindex);
    int dain = thx + pooledWidth*pooledHeight*(ut + bestindex*numunit +  ntr*numunit*numpiece);
    atomicAdd(derData + dain, derPooled[pooledIndex]) ;
    //derData[dain] = derPooled[pooledIndex];
  }
}

template<> vl::Error
vl::impl::maxout_backward<vl::GPU, float>(float* derData,
                                               float const* data,
                                               float const* derPooled,
                                               size_t height, size_t width, size_t depth,
                                              size_t numunit, size_t numpiece)
{
  int pooledWidth = width;
  int pooledHeight = height;
  int pooledVolume = pooledWidth * pooledHeight * depth /  numpiece;

  maxout_backward_kernel<float>
  <<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (derData, data, derPooled,
       pooledHeight, pooledWidth, pooledVolume,
		numunit, numpiece);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}
