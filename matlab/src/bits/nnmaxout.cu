// @file nnpooling.cu
// @brief Pooling block
// @author Andrea Vedaldi

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnmaxout.hpp"
#include "impl/maxout.hpp"

#if ENABLE_GPU
#include "datacu.hpp"
#endif

#include <assert.h>

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                                nnpooling_forward */
/* ---------------------------------------------------------------- */

Error
vl::nnmaxout_forward(vl::Context& context,
                      vl::Tensor output,
                      vl::Tensor data,
                      int numunit, int numpiece)
{
  Error status = vlSuccess ;
	switch (output.getMemoryType()) {
    default:
      assert(false) ;
      return vl::vlErrorUnknown ;
    
#ifdef ENABLE_GPU
    case vl::GPU:
          status = vl::impl::maxout_forward<GPU,float>
          ((float*)output.getMemory(), (float const*)data.getMemory(),
           data.getHeight(), data.getWidth(), data.getDepth() * data.getSize(),
           numunit, numpiece);


      if (status == vlErrorCuda) {
        context.setError(context.getCudaHelper().catchCudaError("maxout_*_forward")) ;
      }
#endif
   }
  return context.passError(status, "maxout_forward: ") ;
}

/* ---------------------------------------------------------------- */
/*                                               nnpooling_backward */
/* ---------------------------------------------------------------- */

Error
vl::nnmaxout_backward(Context& context,
                       Tensor derData,
                       Tensor data,
                       Tensor derPooled,
                       int numunit, int numpiece)
{
  vl::Error status = vlSuccess ;
	switch (derData.getMemoryType()) {
      default:
      assert(false) ;
      return vl::vlErrorUnknown ;
#if ENABLE_GPU
    case vl::GPU:
          status = vl::impl::maxout_backward<GPU,float>
          ((float*)derData.getMemory(), (float const*)data.getMemory(), (float const*)derPooled.getMemory(),
           derData.getHeight(), derData.getWidth(), derData.getDepth() * derData.getSize(),
           numunit, numpiece) ;

      if (status == vlErrorCuda) {
        context.setError(context.getCudaHelper().catchCudaError("maxout*_backward: ")) ;
      }
#endif
	}
  return context.passError(status, "maxout_backward: ") ;
}
