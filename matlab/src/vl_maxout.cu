// @file vl_nnpool.cu
// @brief Pooling block MEX wrapper
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnmaxout.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_SIZE, IN_DEROUTPUT=3, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int numunit ;
  int numpiece ;
  bool backMode = false ;

  int verbosity = 0 ;
  int opt ;


  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 2) { mexErrMsgTxt("The arguments are less than two.") ;}

  if (nin < 4) {
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  vl::MexTensor data(context) ;
  vl::MexTensor derOutput(context) ;

  data.init(in[IN_DATA]) ;
  if (backMode) { derOutput.init(in[IN_DEROUTPUT]) ; }

  if (backMode && ! vl::areCompatible(data, derOutput)) {
    mexErrMsgTxt("DATA and DEROUTPUT are not both CPU or GPU arrays.") ;
  }

  if (!vlmxIsPlainMatrix(in[IN_SIZE],-1,-1)) {
    mexErrMsgTxt("SIZE is not a plain matrix.") ;
  }

      numunit =  mxGetPr(in[1])[0] ;
      numpiece =  mxGetPr(in[2])[0] ;


  /* Basic compatibility of geometry */
   //mexPrintf("input data: %d ",data[0]);
 // mexPrintf("numpiece:%d ",numpiece);
  //mexPrintf("data depth:%d ",data.getDepth());
  if (numunit * numpiece != data.getDepth()) {
    mexErrMsgTxt("hidden unit not equal to maxout layer.") ;
  }

  /* Get the output geometry */
  vl::TensorGeometry outputGeom(data.getHeight(),
                                data.getWidth(),
                                numunit,
                                data.getSize()) ;
   //mexPrintf("input data: %d %d %d %d",data.getHeight(), data.getWidth(),data.getDepth(),data.getSize());

  if (backMode && (derOutput != outputGeom)) {
    mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and POOL.") ;
  }

  /* Create output buffers */
  vl::Device type = data.getMemoryType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;
  vl::MexTensor derFilters(context) ;
  vl::MexTensor derBiases(context) ;

  if (!backMode) {
    output.init(type, outputGeom, 0) ;
  } else {
    derData.init(type, data.getGeometry(), 0) ;
  }

  
  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::Error error ;
  if (!backMode) {
    error = vl::nnmaxout_forward(context,
                                  output, data,
                                  numunit, numpiece);
  } else {
    error = vl::nnmaxout_backward(context,
                                   derData, data, derOutput,
								   numunit, numpiece);
  }

  /* -------------------------------------------------------------- */
  /*                                                         Finish */
  /* -------------------------------------------------------------- */

  if (error != vl::vlSuccess) {
    mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    out[OUT_RESULT] = derData.relinquish() ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
